#include "hip/hip_runtime.h"
#include "../fc2d_cudaclaw.h"

#include "../fc2d_cudaclaw_cuda.h"

#include "cudaclaw_allocate.h"  /* Needed for def of cudaclaw_fluxes_t */


#include <fclaw2d_patch.h>
#include <fclaw2d_global.h>
#include <fclaw2d_vtable.h>

#include <fclaw2d_clawpatch.h>
#include <fclaw2d_clawpatch_options.h>
#include <fc2d_cudaclaw_options.h>

#include "../fc2d_cudaclaw_check.h"  /* CHECK defined here */

#include <fc2d_cuda_profiler.h>
#include <hipcub/hipcub.hpp>

#define thread_count 224


/* Put header here so it doesn't have to go in *.h file */
__global__
void cudaclaw_flux2_and_update_batch (const int mx,    const int my, 
                                      const int meqn,  const int mbc, 
                                      const int maux,  const int mwaves, 
                                      const int mwork,
                                      const double dt, const double t,
                                      struct cudaclaw_fluxes* array_fluxes_struct_dev,
                                      double * maxcflblocks,
                                      cudaclaw_cuda_rpn2_t rpn2,
                                      cudaclaw_cuda_rpt2_t rpt2,
                                      cudaclaw_cuda_b4step2_t b4step2);

__global__
void cudaclaw_compute_speeds_batch (const int mx,    const int my, 
                                    const int meqn,  const int mbc, 
                                    const int maux,  const int mwaves, 
                                    const int mwork,
                                    const double dt, const double t,
                                    cudaclaw_fluxes_t* array_fluxes_struct,
                                    double * maxcflblocks,
                                    cudaclaw_cuda_speeds_t compute_speeds,
                                    cudaclaw_cuda_b4step2_t b4step2);


double cudaclaw_step2_batch(fclaw2d_global_t *glob,
        cudaclaw_fluxes_t* array_fluxes_struct, 
        int batch_size, double t, double dt)
{
    PROFILE_CUDA_GROUP("cudaclaw_step2_batch",1);

    size_t size, bytes, bytes_per_thread;
    float bytes_kb;
    int I_q, I_aux, mwork;
    int i;

    int mx,my,mbc,maux,meqn,mwaves;
    double maxcfl;

    double* maxcflblocks_dev;    

    double *membuffer_cpu, *membuffer_dev;
    cudaclaw_fluxes_t *array_fluxes_struct_dev;

    cudaclaw_fluxes_t* fluxes;

    /* To get patch-independent parameters */
    fc2d_cudaclaw_options_t *clawopt;
    fclaw2d_clawpatch_options_t *clawpatch_opt;

    /* ---------------------------------- start code ---------------------------------- */
    FCLAW_ASSERT(batch_size > 0);

    clawopt = fc2d_cudaclaw_get_options(glob);
    mwaves = clawopt->mwaves;

    fc2d_cudaclaw_vtable_t*  cuclaw_vt = fc2d_cudaclaw_vt(glob);
    FCLAW_ASSERT(cuclaw_vt->cuda_rpn2 != NULL);
    if (clawopt->order[1] > 0)
    {
        FCLAW_ASSERT(cuclaw_vt->cuda_rpt2 != NULL);        
    }

    clawpatch_opt = fclaw2d_clawpatch_get_options(glob);
    mx = clawpatch_opt->mx;
    my = clawpatch_opt->my;
    mbc = clawpatch_opt->mbc;
    maux = clawpatch_opt->maux;
    meqn = clawpatch_opt->meqn;  

    fluxes = &(array_fluxes_struct[0]);
    size = batch_size*(fluxes->num + fluxes->num_aux);
    bytes = size*sizeof(double);

    /* ---------------------------------- Merge Memory ---------------------------------*/ 
    membuffer_cpu = cudaclaw_get_cpu_membuffer();
    membuffer_dev = cudaclaw_get_gpu_membuffer();
    fclaw2d_timer_start_threadsafe (&glob->timers[FCLAW2D_TIMER_CUDA_MEMCOPY_H2H]);       
    {
        PROFILE_CUDA_GROUP("Copy data on patches to CPU memory buffer",5);    
        for(i = 0; i < batch_size; i++)   
        {
            fluxes = &(array_fluxes_struct[i]);    

            I_q = i*fluxes->num;
            memcpy(&membuffer_cpu[I_q]  ,fluxes->qold ,fluxes->num_bytes);
            fluxes->qold_dev = &membuffer_dev[I_q];

            if (fluxes->num_aux > 0)
            {
                I_aux = batch_size*fluxes->num + i*fluxes->num_aux;
                memcpy(&membuffer_cpu[I_aux],fluxes->aux  ,fluxes->num_bytes_aux);                
                fluxes->aux_dev  = &membuffer_dev[I_aux];
            }
        }  
    }     
    fclaw2d_timer_stop_threadsafe(&glob->timers[FCLAW2D_TIMER_CUDA_MEMCOPY_H2H]);       
  
    

    fclaw2d_timer_start_threadsafe(&glob->timers[FCLAW2D_TIMER_CUDA_MEMCOPY_H2D]);       
    {
        PROFILE_CUDA_GROUP("Copy CPU buffer to device memory",3);              
        CHECK(hipMemcpy(membuffer_dev, membuffer_cpu, bytes, hipMemcpyHostToDevice));            
    }            
    fclaw2d_timer_stop_threadsafe(&glob->timers[FCLAW2D_TIMER_CUDA_MEMCOPY_H2D]);       


    /* -------------------------------- Work with array --------------------------------*/ 

    {
        PROFILE_CUDA_GROUP("Copy fluxes to device memory",3);    

        array_fluxes_struct_dev = cudaclaw_get_flux_buffer();

        CHECK(hipMemcpy(array_fluxes_struct_dev, array_fluxes_struct, 
                         batch_size*sizeof(cudaclaw_fluxes_t), 
                         hipMemcpyHostToDevice));
    }        

    {
        PROFILE_CUDA_GROUP("Malloc for CFL computation",2);    

        /* Data needed to reduce CFL number */
        CHECK(hipMalloc(&maxcflblocks_dev,batch_size*sizeof(double)));         
    }


#if 0
    {
        PROFILE_CUDA_GROUP("Configure and call to compute speeds",6);  

        /* Compute speeds */
        int block_size = FC2D_CUDACLAW_BLOCK_SIZE;

        dim3 block(block_size,1,1);
        dim3 grid(1,1,batch_size);

        mwork = 2*(meqn + maux) + mwaves;
        bytes_per_thread = sizeof(double)*mwork;
        bytes = bytes_per_thread*block_size;
        bytes_kb = bytes/1024.0;

        cudaclaw_compute_speeds_batch <<<grid,block,bytes>>>(mx,my,meqn, mbc, maux, mwaves,
                                                             mwork, dt, t, 
                                                             array_fluxes_struct_dev,
                                                             maxcflblocks_dev,
                                                             cuclaw_vt->cuda_speeds,
                                                             cuclaw_vt->cuda_b4step2);

        hipDeviceSynchronize();

        hipError_t code = hipPeekAtLastError();
        if (code != hipSuccess) 
        {
            fclaw_global_essentialf("ERROR (cudaclaw_step2.cu (compute_speeds)) : %s\n\n", 
                                    hipGetErrorString(code));
            exit(code);
        }
    }        
#endif    

    {
        PROFILE_CUDA_GROUP("Configure and call main kernel",6);  

        /* Determine shared memory size */
        int block_size = FC2D_CUDACLAW_BLOCK_SIZE;
        //int block_size = thread_count;
        dim3 block(block_size,1,1);
        dim3 grid(1,1,batch_size);

        int mwork1 = 4*meqn + 2*maux + mwaves + meqn*mwaves;
        int mwork2 = 5*meqn + 6*maux;
        mwork = (mwork1 > mwork2) ? mwork1 : mwork2;
        bytes_per_thread = sizeof(double)*mwork;
        bytes = bytes_per_thread*block_size;

        bytes_kb = bytes/1024.0;
        //fclaw_global_essentialf("[fclaw] Shared memory  : %0.2f kb\n\n",bytes_kb);

        cudaclaw_flux2_and_update_batch<<<grid,block,bytes>>>(mx,my,meqn,mbc,maux,mwaves,
                                                              mwork, dt,t,
                                                              array_fluxes_struct_dev,
                                                              maxcflblocks_dev,
                                                              cuclaw_vt->cuda_rpn2,
                                                              cuclaw_vt->cuda_rpt2,
                                                              cuclaw_vt->cuda_b4step2);
        hipDeviceSynchronize();

        
        hipError_t code = hipPeekAtLastError();

        if (code != hipSuccess) 
        {
            fclaw_global_essentialf("ERROR (cudaclaw_step2.cu) : %s\n", 
                                    hipGetErrorString(code));
            exit(code);
        }
    }

    /* -------------------------------- Finish CFL ------------------------------------*/ 
    {
        PROFILE_CUDA_GROUP("Finish CFL",2);
        void    *temp_storage_dev = NULL;
        size_t  temp_storage_bytes = 0;
        double  *cflgrid_dev;

        hipMalloc(&cflgrid_dev, sizeof(double));  
        CubDebugExit(hipcub::DeviceReduce::Max(temp_storage_dev,temp_storage_bytes,
                                            maxcflblocks_dev,cflgrid_dev,batch_size));
        hipMalloc(&temp_storage_dev, temp_storage_bytes);
        CubDebugExit(hipcub::DeviceReduce::Max(temp_storage_dev,temp_storage_bytes,
                                            maxcflblocks_dev,cflgrid_dev,batch_size));
        hipMemcpy(&maxcfl, cflgrid_dev, sizeof(double),hipMemcpyDeviceToHost);
        hipFree(temp_storage_dev);
        hipFree(cflgrid_dev);
    }


	
    /* -------------------------- Copy q back to host ----------------------------------*/ 
    fclaw2d_timer_start_threadsafe (&glob->timers[FCLAW2D_TIMER_CUDA_MEMCOPY_D2H]);       

    {
        PROFILE_CUDA_GROUP("Copy device memory buffer back to CPU",3);

        CHECK(hipMemcpy(membuffer_cpu, membuffer_dev, batch_size*fluxes->num_bytes, 
                         hipMemcpyDeviceToHost));
    }
    fclaw2d_timer_stop_threadsafe (&glob->timers[FCLAW2D_TIMER_CUDA_MEMCOPY_D2H]);       

    fclaw2d_timer_start_threadsafe (&glob->timers[FCLAW2D_TIMER_CUDA_MEMCOPY_H2H]);       
    {
        PROFILE_CUDA_GROUP("Copy CPU buffer back to patches",5);
        for (i = 0; i < batch_size; ++i)    
        {      
            fluxes = &(array_fluxes_struct[i]);
            I_q = i*fluxes->num;

            memcpy(fluxes->qold,&membuffer_cpu[I_q],fluxes->num_bytes);
        }        
    }
    fclaw2d_timer_stop_threadsafe (&glob->timers[FCLAW2D_TIMER_CUDA_MEMCOPY_H2H]);       

    return maxcfl;
}

